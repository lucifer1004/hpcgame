
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

void run(const char *command) {
  FILE *fp = popen(command, "r");
  char path[PATH_MAX];
  while (fgets(path, PATH_MAX, fp) != NULL)
    printf("%s", path);
  int status = pclose(fp);
  if (status != 0) {
    printf("Error: %d\n", status);
    exit(status);
  }
}

int main() {
  run("module load pyenv");

  FILE *py;
  if ((py = fopen("./out.py", "w"))) {
    fprintf(py, "%s\n", R"V0G0N(
import numpy as np
import taichi as ti
import struct

ti.init(arch=ti.gpu, default_fp=ti.f32)

with open("conf.data", mode="rb") as bfile:
    contents = bfile.read()
    ray_density = struct.unpack("<i", contents[:4])[0]
    sensor_density = struct.unpack("<i", contents[4:8])[0]
    pane_count = struct.unpack("<i", contents[8:12])[0]
    pane_pos = np.zeros(pane_count)
    ptr = 12
    for i in range(pane_count):
        pane_pos[i] = struct.unpack("<f", contents[ptr:ptr+4])[0]
        ptr += 4
    lens_index = np.zeros(pane_count + 1, dtype=np.int32)
    for i in range(pane_count + 1):
        lens_index[i] = struct.unpack("<i", contents[ptr:ptr+4])[0]
        ptr += 4
    lens = np.zeros(4 * lens_index[-1])
    for i in range(0, 4 * lens_index[-1], 4):
        lens[i] = struct.unpack("<f", contents[ptr:ptr+4])[0]
        lens[i + 1] = struct.unpack("<f", contents[ptr+4:ptr+8])[0]
        lens[i + 2] = struct.unpack("<f", contents[ptr+8:ptr+12])[0]
        lens[i + 3] = struct.unpack("<f", contents[ptr+12:ptr+16])[0]
        ptr += 16

sensor_data = ti.field(dtype=ti.i32, shape=(sensor_density, sensor_density))
lens_data = ti.field(dtype=ti.f32, shape=len(lens))
lens_data.from_numpy(lens)
pos = ti.field(dtype=ti.f32, shape=(pane_count))
pos.from_numpy(pane_pos)
index = ti.field(dtype=ti.i32, shape=(pane_count + 1))
index.from_numpy(lens_index)


@ti.kernel
def causcal():
    rs = 1.0 / ray_density

    ti.loop_config(block_dim=128)
    ti.block_local(lens_data)
    ti.block_local(pos)
    ti.block_local(index)
    for i, j in ti.ndrange(ray_density, ray_density):
        y = (i + 0.5) * rs
        x = (j + 0.5) * rs
        z = 0.0
        kx = 0.0
        ky = 0.0
        invkz = 1.0
        broken = False

        for k in range(pane_count):
            x += (pos[k] - z) * kx * invkz
            y += (pos[k] - z) * ky * invkz

            if 0.0 < x < 1.0 and 0.0 < y < 1.0:
                z = pos[k]
                gx = 0.0
                gy = 0.0
                s = index[k]
                e = index[k + 1]
                for l in range(s, e):
                    rx = x - lens_data[4 * l]
                    ry = y - lens_data[4 * l + 1]
                    rr = rx * rx + ry * ry
                    invr02 = lens_data[4 * l + 2] ** 2
                    if rr < invr02 * 0.99999:
                        invr02 = 1.0 / invr02
                        rr *= invr02
                        rr = 4.0 / (1.0 - rr)
                        rr = -0.5 * ti.exp(4.0 - rr) * rr * \
                            rr * invr02 * lens_data[4 * l + 3]
                        gx += rx * rr
                        gy += ry * rr

                kx += gx
                ky += gy
                kp = kx * kx + ky * ky
                if kp < 1.0:
                    invkz = 1.0 / ti.sqrt(1.0 - kp)
                else:
                    broken = True
                    break
            else:
                broken = True
                break

        if not broken:
            x += (1.0 - z) * kx * invkz
            y += (1.0 - z) * ky * invkz

            if 0.0 < x < 1.0 and 0.0 < y < 1.0:
                sensor_data[int(y * sensor_density),
                            int(x * sensor_density)] += 1


causcal()
out = sensor_data.to_numpy()

with open("out.data", mode="wb") as bfile:
    for i in range(sensor_density):
        for j in range(sensor_density):
            bfile.write(struct.pack("<i", out[i, j]))
)V0G0N");
    fclose(py);
  }

  run("/data/software/tools/anaconda/3/envs/pyenv/bin/python3 ./out.py");
  return 0;
}