
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define GRID_DIM 512
#define BLOCK_DIM 512

struct pane_t {
  int panecount;
  float *panepos;
  int *lensindex;
  float *lensdata;
};

struct result_t {
  int raydensity;
  int sensordensity;
  int *sensordata;
};

int *cudata, *lensindex;
float *panepos, *lensdata;

__global__ void causcal_kernel(int *cudata, int *lensindex, float *panepos,
                               float *lensdata, int raydensity,
                               int sensordensity, int panecount) {
  float rs = 1.0f / raydensity;
  float y, x, z, kx, ky, invkz;

  for (int i = blockIdx.x; i < raydensity; i += GRID_DIM) {
    for (int j = threadIdx.x; j < raydensity; j += BLOCK_DIM) {
      y = rs * (0.5f + i);
      x = rs * (0.5f + j);
      z = 0.0f;
      kx = 0.0f;
      ky = 0.0f;
      invkz = 1.0f;
      bool broken = false;

      for (int k = 0; k < panecount; k++) {
        x += (panepos[k] - z) * kx * invkz;
        y += (panepos[k] - z) * ky * invkz;
        if (x > 0.0f && x < 1.0f && y > 0.0f && y < 1.0f) {
          z = panepos[k];

          float gx = 0.0f, gy = 0.0f;
          int is = lensindex[k];
          int ie = lensindex[k + 1];
          for (int l = 4 * is; l < 4 * ie; l += 4) {
            float rx = x - lensdata[l];
            float ry = y - lensdata[l + 1];
            float r = rx * rx + ry * ry;
            float invr02 = lensdata[l + 2] * lensdata[l + 2];
            if (r < invr02 * 0.99999f) {
              invr02 = 1.0f / invr02;
              r = r * invr02;
              r = 4.0f / (1.0f - r);
              r = -0.5f * exp(4.0f - r) * r * r * invr02 * lensdata[l + 3];
              gx += r * rx;
              gy += r * ry;
            }
          }

          kx += gx;
          ky += gy;
          float kp = kx * kx + ky * ky;
          if (kp < 1.0f) {
            invkz = 1.0f / sqrt(1.0f - kp);
          } else {
            broken = true;
            break;
          }
        } else {
          broken = true;
          break;
        }
      }

      if (!broken) {
        x += (1.0f - z) * kx * invkz;
        y += (1.0f - z) * ky * invkz;

        if (x > 0.0f && x < 1.0f && y > 0.0f && y < 1.0f) {
          int pixelindex =
              int(x * sensordensity) + sensordensity * int(y * sensordensity);
          atomicAdd(&cudata[pixelindex], 1);
        }
      }
    }
  }
}

void causcal_cuda(pane_t pane, result_t result) {
  hipMalloc((void **)&cudata,
             sizeof(int) * result.sensordensity * result.sensordensity);

  hipMalloc((void **)&panepos, sizeof(float) * pane.panecount);
  hipMemcpy(panepos, pane.panepos, sizeof(float) * pane.panecount,
             hipMemcpyHostToDevice);

  hipMalloc((void **)&lensindex, sizeof(int) * (pane.panecount + 1));
  hipMemcpy(lensindex, pane.lensindex, sizeof(int) * (pane.panecount + 1),
             hipMemcpyHostToDevice);

  hipMalloc((void **)&lensdata,
             sizeof(float) * pane.lensindex[pane.panecount] * 4);
  hipMemcpy(lensdata, pane.lensdata,
             sizeof(float) * pane.lensindex[pane.panecount] * 4,
             hipMemcpyHostToDevice);

  causcal_kernel<<<GRID_DIM, BLOCK_DIM>>>(cudata, lensindex, panepos, lensdata,
                                          result.raydensity,
                                          result.sensordensity, pane.panecount);

  memset(result.sensordata, 0,
         result.sensordensity * result.sensordensity * sizeof(int));

  hipMemcpy(result.sensordata, cudata,
             sizeof(int) * result.sensordensity * result.sensordensity,
             hipMemcpyDeviceToHost);
}

int loadconf(const char *fn, pane_t &pane, result_t &result) {
  size_t n;
  FILE *fi;
  if ((fi = fopen(fn, "rb"))) {
    n = fread(&result.raydensity, 4, 1, fi);
    n = fread(&result.sensordensity, 4, 1, fi);
    result.sensordata = (int *)malloc(result.sensordensity *
                                      result.sensordensity * sizeof(int));

    n = fread(&pane.panecount, 4, 1, fi);
    pane.panepos = (float *)malloc(pane.panecount * sizeof(float));
    n = fread(pane.panepos, 4, pane.panecount, fi);
    pane.lensindex = (int *)malloc((pane.panecount + 1) * sizeof(int));
    n = fread(pane.lensindex, 4, pane.panecount + 1, fi);
    pane.lensdata =
        (float *)malloc(pane.lensindex[pane.panecount] * 4 * sizeof(float));
    n = fread(pane.lensdata, 4, pane.lensindex[pane.panecount] * 4, fi);

    fclose(fi);
  }
  printf("%lu\n", n);
  return 0;
}

int main() {
  result_t result;
  pane_t pane;
  loadconf("./conf.data", pane, result);

  causcal_cuda(pane, result);

  FILE *fi;
  if ((fi = fopen("./out.data", "wb"))) {
    fwrite(result.sensordata, 1,
           result.sensordensity * result.sensordensity * sizeof(int), fi);
    fclose(fi);
  }

  return 0;
}
