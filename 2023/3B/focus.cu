
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define GRID_DIM 512
#define BLOCK_DIM 512
#define N 512

struct pane_t {
  int panecount;
  float *panepos;
  int *lensindex;
  float *lensdata;
};

struct result_t {
  int raydensity;
  int sensordensity;
  int *sensordata;
};

int *cudata, *lensindex;
float *panepos, *lensdata;

__global__ void causcal_kernel(int *cudata, int *lensindex, float *panepos,
                               float *lensdata, int raydensity,
                               int sensordensity, int panecount) {
  float rs = 1.0f / raydensity;
  float y, x, z, kx, ky, invkz;

  int i_start = blockIdx.x * raydensity / GRID_DIM;
  int j_start = threadIdx.x * raydensity / BLOCK_DIM;
  float x_min = rs * (0.5f + j_start),
        x_max = rs * (0.5f + j_start + (float)raydensity / BLOCK_DIM - 1);
  float y_min = rs * (0.5f + i_start),
        y_max = rs * (0.5f + i_start + (float)raydensity / GRID_DIM - 1);

  int intersected[N];
  int ptr = 0;
  for (int l = 0; l < panecount * 256 * 4; l += 4) {
    float rx = lensdata[l];
    float ry = lensdata[l + 1];
    float r = lensdata[l + 2];
    if (rx - r < x_max && rx + r > x_min && ry - r < y_max && ry + r > y_min)
      intersected[ptr++] = l;
  }
  intersected[ptr++] = 10000000;

  for (int i = 0; i < raydensity / GRID_DIM; i++) {
    for (int j = 0; j < raydensity / BLOCK_DIM; j++) {
      y = rs * (0.5f + i_start + i);
      x = rs * (0.5f + j_start + j);
      z = 0.0f;
      kx = 0.0f;
      ky = 0.0f;
      invkz = 1.0f;
      bool broken = false;

      int pptr = 0;
      for (int k = 0; k < panecount; k++) {
        x += (panepos[k] - z) * kx * invkz;
        y += (panepos[k] - z) * ky * invkz;
        if (x > 0.0f && x < 1.0f && y > 0.0f && y < 1.0f) {
          z = panepos[k];

          float gx = 0.0f, gy = 0.0f;
          int is = lensindex[k];
          int ie = lensindex[k + 1];
          for (; pptr < ptr && intersected[pptr] >= is * 4 &&
                 intersected[pptr] < ie * 4;
               pptr++) {
            int l = intersected[pptr];
            float rx = x - lensdata[l];
            float ry = y - lensdata[l + 1];
            float r = rx * rx + ry * ry;
            float invr02 = lensdata[l + 2] * lensdata[l + 2];
            if (r < invr02 * 0.99999f) {
              invr02 = 1.0f / invr02;
              r = r * invr02;
              r = 4.0f / (1.0f - r);
              r = -0.5f * exp(4.0f - r) * r * r * invr02 * lensdata[l + 3];
              gx += r * rx;
              gy += r * ry;
            }
          }

          kx += gx;
          ky += gy;
          float kp = kx * kx + ky * ky;
          if (kp < 1.0f) {
            invkz = 1.0f / sqrt(1.0f - kp);
          } else {
            broken = true;
            break;
          }
        } else {
          broken = true;
          break;
        }
      }

      if (!broken) {
        x += (1.0f - z) * kx * invkz;
        y += (1.0f - z) * ky * invkz;

        if (x > 0.0f && x < 1.0f && y > 0.0f && y < 1.0f) {
          int pixelindex =
              int(x * sensordensity) + sensordensity * int(y * sensordensity);
          atomicAdd(&cudata[pixelindex], 1);
        }
      }
    }
  }
}

void causcal_cuda(pane_t pane, result_t result) {
  hipMalloc((void **)&cudata,
             sizeof(int) * result.sensordensity * result.sensordensity);

  hipMalloc((void **)&panepos, sizeof(float) * pane.panecount);
  hipMemcpy(panepos, pane.panepos, sizeof(float) * pane.panecount,
             hipMemcpyHostToDevice);

  hipMalloc((void **)&lensindex, sizeof(int) * (pane.panecount + 1));
  hipMemcpy(lensindex, pane.lensindex, sizeof(int) * (pane.panecount + 1),
             hipMemcpyHostToDevice);

  hipMalloc((void **)&lensdata,
             sizeof(float) * pane.lensindex[pane.panecount] * 4);
  hipMemcpy(lensdata, pane.lensdata,
             sizeof(float) * pane.lensindex[pane.panecount] * 4,
             hipMemcpyHostToDevice);

  causcal_kernel<<<GRID_DIM, BLOCK_DIM>>>(cudata, lensindex, panepos, lensdata,
                                          result.raydensity,
                                          result.sensordensity, pane.panecount);

  memset(result.sensordata, 0,
         result.sensordensity * result.sensordensity * sizeof(int));

  hipMemcpy(result.sensordata, cudata,
             sizeof(int) * result.sensordensity * result.sensordensity,
             hipMemcpyDeviceToHost);
}

int loadconf(const char *fn, pane_t &pane, result_t &result) {
  size_t n;
  FILE *fi;
  if ((fi = fopen(fn, "rb"))) {
    n = fread(&result.raydensity, 4, 1, fi);
    n = fread(&result.sensordensity, 4, 1, fi);
    result.sensordata = (int *)malloc(result.sensordensity *
                                      result.sensordensity * sizeof(int));

    n = fread(&pane.panecount, 4, 1, fi);
    pane.panepos = (float *)malloc(pane.panecount * sizeof(float));
    n = fread(pane.panepos, 4, pane.panecount, fi);
    pane.lensindex = (int *)malloc((pane.panecount + 1) * sizeof(int));
    n = fread(pane.lensindex, 4, pane.panecount + 1, fi);
    pane.lensdata =
        (float *)malloc(pane.lensindex[pane.panecount] * 4 * sizeof(float));
    n = fread(pane.lensdata, 4, pane.lensindex[pane.panecount] * 4, fi);

    fclose(fi);
  }
  printf("%lu\n", n);
  return 0;
}

int main() {
  result_t result;
  pane_t pane;
  loadconf("./conf.data", pane, result);

  causcal_cuda(pane, result);

  FILE *fi;
  if ((fi = fopen("./out.data", "wb"))) {
    fwrite(result.sensordata, 1,
           result.sensordensity * result.sensordensity * sizeof(int), fi);
    fclose(fi);
  }

  return 0;
}
