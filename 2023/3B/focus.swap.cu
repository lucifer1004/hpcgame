
#include <hip/hip_runtime.h>
#include <ctime>
#include <malloc.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define MAXIMUM_PANES 32
#define TEMPLATE_PER_PANE 256
#define GRID_DIM 256
#define BLOCK_DIM 256

struct pane_t {
  int panecount;
  float *panepos;
  int *lensindex;
  float *lensdata;
};

struct result_t {
  int raydensity;
  int sensordensity;
  int *sensordata;
};

int *cudata, *lensindex;
float *panepos, *lensdata;

#define MAKE_CAUSCAL_KERNEL(N)                                                 \
  __global__ void causcal_kernel_##N(int *cudata, float *panepos,              \
                                     float *lensdata, int raydensity,          \
                                     int sensordensity, int panecount) {       \
    float rs = 1.0f / raydensity;                                              \
    float y, x, z, kx, ky, invkz;                                              \
    __shared__ float panepos_[MAXIMUM_PANES];                                  \
                                                                               \
    if (threadIdx.x < panecount) {                                             \
      panepos_[threadIdx.x] = panepos[threadIdx.x];                            \
    }                                                                          \
    __syncthreads();                                                           \
                                                                               \
    __shared__ float lensdata_[TEMPLATE_PER_PANE * 4];                         \
                                                                               \
    float xx[N * N], yy[N * N], kxx[N * N], kyy[N * N];                        \
    bool flag[N * N];                                                          \
    for (int i = 0; i < N; i++) {                                              \
      for (int j = 0; j < N; j++) {                                            \
        xx[i * N + j] = rs * (0.5f + blockIdx.x + i * GRID_DIM);               \
        yy[i * N + j] = rs * (0.5f + threadIdx.x + j * BLOCK_DIM);             \
        kxx[i * N + j] = 0.0f;                                                 \
        kyy[i * N + j] = 0.0f;                                                 \
        flag[i * N + j] = false;                                               \
      }                                                                        \
    }                                                                          \
                                                                               \
    for (int k = 0; k < panecount; k++) {                                      \
      for (int j = threadIdx.x; j < TEMPLATE_PER_PANE; j += BLOCK_DIM) {       \
        lensdata_[j * 4] = lensdata[k * TEMPLATE_PER_PANE * 4 + j * 4];        \
        lensdata_[j * 4 + 1] =                                                 \
            lensdata[k * TEMPLATE_PER_PANE * 4 + j * 4 + 1];                   \
        lensdata_[j * 4 + 2] =                                                 \
            lensdata[k * TEMPLATE_PER_PANE * 4 + j * 4 + 2] *                  \
            lensdata[k * TEMPLATE_PER_PANE * 4 + j * 4 + 2];                   \
        lensdata_[j * 4 + 3] =                                                 \
            lensdata[k * TEMPLATE_PER_PANE * 4 + j * 4 + 3];                   \
      }                                                                        \
      __syncthreads();                                                         \
                                                                               \
      for (int i = 0; i < N; i++) {                                            \
        for (int j = 0; j < N; j++) {                                          \
          if (!flag[i * N + j]) {                                              \
            kx = kxx[i * N + j];                                               \
            ky = kyy[i * N + j];                                               \
            invkz = 1.0f / sqrt(1.0f - kx * kx - ky * ky);                     \
            x = xx[i * N + j] + (panepos_[k] - z) * kx * invkz;                \
            y = yy[i * N + j] + (panepos_[k] - z) * ky * invkz;                \
            if (x <= 0.0f || x >= 1.0f || y <= 0.0f || y >= 1.0f) {            \
              flag[i * N + j] = true;                                          \
              break;                                                           \
            }                                                                  \
                                                                               \
            if (!flag[i * N + j]) {                                            \
              float gx = 0.0f, gy = 0.0f;                                      \
              for (int l = 0; l < TEMPLATE_PER_PANE * 4; l += 4) {             \
                float rx = x - lensdata_[l];                                   \
                float ry = y - lensdata_[l + 1];                               \
                float r = rx * rx + ry * ry;                                   \
                float invr02 = lensdata_[l + 2];                               \
                if (r < invr02 * 0.99999f) {                                   \
                  invr02 = 1.0f / invr02;                                      \
                  r = r * invr02;                                              \
                  r = 4.0f / (1.0f - r);                                       \
                  r = -0.5f * exp(4.0f - r) * r * r * invr02 *                 \
                      lensdata_[l + 3];                                        \
                  gx += r * rx;                                                \
                  gy += r * ry;                                                \
                }                                                              \
              }                                                                \
                                                                               \
              kx += gx;                                                        \
              ky += gy;                                                        \
              float kp = kx * kx + ky * ky;                                    \
              if (kp >= 1.0f) {                                                \
                flag[i * N + j] = true;                                        \
                break;                                                         \
              }                                                                \
            }                                                                  \
          }                                                                    \
                                                                               \
          xx[i * N + j] = x;                                                   \
          yy[i * N + j] = y;                                                   \
          kxx[i * N + j] = kx;                                                 \
          kyy[i * N + j] = ky;                                                 \
        }                                                                      \
      }                                                                        \
                                                                               \
      z = panepos_[k];                                                         \
    }                                                                          \
                                                                               \
    for (int i = 0; i < N; i++) {                                              \
      for (int j = 0; j < N; j++) {                                            \
        if (!flag[i * N + j]) {                                                \
          kx = kxx[i * N + j];                                                 \
          ky = kyy[i * N + j];                                                 \
          invkz = 1.0f / sqrt(1.0f - kx * kx - ky * ky);                       \
          x = xx[i * N + j] + (1.0f - z) * kx * invkz;                         \
          y = yy[i * N + j] + (1.0f - z) * ky * invkz;                         \
          if (x > 0.0f && x < 1.0f && y > 0.0f && y < 1.0f) {                  \
            int pixelindex = int(x * sensordensity) +                          \
                             sensordensity * int(y * sensordensity);           \
            atomicAdd(&cudata[pixelindex], 1);                                 \
          }                                                                    \
        }                                                                      \
      }                                                                        \
    }                                                                          \
  }

MAKE_CAUSCAL_KERNEL(16)
MAKE_CAUSCAL_KERNEL(64)

void causcal_cuda(pane_t pane, result_t result) {
  hipMalloc((void **)&cudata,
             sizeof(int) * result.sensordensity * result.sensordensity);

  hipMalloc((void **)&panepos, sizeof(float) * pane.panecount);
  hipMemcpy(panepos, pane.panepos, sizeof(float) * pane.panecount,
             hipMemcpyHostToDevice);

  hipMalloc((void **)&lensdata,
             sizeof(float) * pane.lensindex[pane.panecount] * 4);
  hipMemcpy(lensdata, pane.lensdata,
             sizeof(float) * pane.lensindex[pane.panecount] * 4,
             hipMemcpyHostToDevice);

  if (result.raydensity == 4096) {
    causcal_kernel_16<<<GRID_DIM, BLOCK_DIM,
                        sizeof(float) *
                            (MAXIMUM_PANES + TEMPLATE_PER_PANE * 4)>>>(
        cudata, panepos, lensdata, result.raydensity, result.sensordensity,
        pane.panecount);
  } else {
    causcal_kernel_64<<<GRID_DIM, BLOCK_DIM,
                        sizeof(float) *
                            (MAXIMUM_PANES + TEMPLATE_PER_PANE * 4)>>>(
        cudata, panepos, lensdata, result.raydensity, result.sensordensity,
        pane.panecount);
  }

  memset(result.sensordata, 0,
         result.sensordensity * result.sensordensity * sizeof(int));

  hipMemcpy(result.sensordata, cudata,
             sizeof(int) * result.sensordensity * result.sensordensity,
             hipMemcpyDeviceToHost);

  hipFree(cudata);
  hipFree(panepos);
  hipFree(lensdata);
}

int loadconf(const char *fn, pane_t &pane, result_t &result) {
  size_t n;
  FILE *fi;
  if ((fi = fopen(fn, "rb"))) {
    n = fread(&result.raydensity, 4, 1, fi);
    n = fread(&result.sensordensity, 4, 1, fi);
    result.sensordata = (int *)malloc(result.sensordensity *
                                      result.sensordensity * sizeof(int));

    n = fread(&pane.panecount, 4, 1, fi);
    pane.panepos = (float *)malloc(pane.panecount * sizeof(float));
    n = fread(pane.panepos, 4, pane.panecount, fi);
    pane.lensindex = (int *)malloc((pane.panecount + 1) * sizeof(int));
    n = fread(pane.lensindex, 4, pane.panecount + 1, fi);
    pane.lensdata =
        (float *)malloc(pane.lensindex[pane.panecount] * 4 * sizeof(float));
    n = fread(pane.lensdata, 4, pane.lensindex[pane.panecount] * 4, fi);

    fclose(fi);
  }
  printf("%lu\n", n);
  return 0;
}

int main() {
  clock_t before = clock();
  result_t result;
  pane_t pane;
  loadconf("./conf.data", pane, result);
  printf("Loading configuration takes %ld ms.\n",
         (clock() - before) * 1000 / CLOCKS_PER_SEC);
  before = clock();

  causcal_cuda(pane, result);
  hipDeviceSynchronize();
  printf("Processing data takes %ld ms.\n",
         (clock() - before) * 1000 / CLOCKS_PER_SEC);
  before = clock();

  FILE *fi;
  if ((fi = fopen("./out.data", "wb"))) {
    fwrite(result.sensordata, 1,
           result.sensordensity * result.sensordensity * sizeof(int), fi);
    fclose(fi);
  }
  printf("Writing output takes %ld ms.\n",
         (clock() - before) * 1000 / CLOCKS_PER_SEC);
  before = clock();

  return 0;
}
